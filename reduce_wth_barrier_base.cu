#include "hip/hip_runtime.h"
// Author: Nic Olsen

#include <iostream>
#include <stdio.h>
#include "reduce.cuh"
#include <hip/hip_cooperative_groups.h>
namespace cg = cooperative_groups;

 __global__ void reduce_kernel(int* g_idata, int* g_odata, unsigned int N) {
    extern __shared__ int sdata[];
  
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    for (unsigned int n = N; n > 1; n = (n + blockDim.x - 1) / blockDim.x){
    if (i < n) {
        sdata[tid] = g_idata[i];
    } else {
        sdata[tid] = 0;
    }

    __syncthreads();

    // Sequential addressing alteration
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Write out reduced portion of the output
    if (tid == 0) {
        g_odata[blockIdx.x] = sdata[0];
        printf("Sum for blockIdx.x is %d and %d\n", blockIdx.x, g_odata[blockIdx.x]);
    }

    cg::grid_group grid = cg::this_grid(); 
    cg::sync(grid);

    int* tmp = g_idata;
    g_idata = g_odata;
    g_odata = tmp;
}

}

__host__ int reduce(const int* arr, unsigned int N, unsigned int threads_per_block) {
    // Workspace NOTE: Could be smaller
    int* a;
    int* b;
    hipMallocManaged(&a, N * sizeof(int));
    hipMallocManaged(&b, N * sizeof(int));
    hipMemcpy(a, arr, N * sizeof(int), hipMemcpyHostToDevice);

    //for (unsigned int n = N; n > 1; n = (n + threads_per_block - 1) / threads_per_block) {
        reduce_kernel<<<(N + threads_per_block - 1) / threads_per_block, threads_per_block,
                        threads_per_block * sizeof(int)>>>(a, b, N);

        // Swap input and output arrays
        //int* tmp = a;
        //a = b;
        //b = tmp;
   // }
    hipDeviceSynchronize();

    int sum = a[0];

    hipFree(a);
    hipFree(b);

    return sum;
}
