// Author: Nic Olsen

#include <hip/hip_runtime.h>
#include <iostream>

#include "reduce.cuh"

int main(int argc, char* argv[]) {
    int N = std::atoi(argv[1]);
    unsigned int threads_per_block = std::atoi(argv[2]);

    int* arr = new int[N];

    for (int i = 0; i < N; i++) {
        arr[i] = 1;
    }
    int correct_sum = N;

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    int sum = reduce(arr, N, threads_per_block);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Get the elapsed time in milliseconds
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    std::cout << "Sum: " << sum << std::endl;
    std::cout << "Correct sum: " << correct_sum << std::endl;
    std::cout << "time (ms) " << ms << std::endl;

    delete[] arr;
    return 0;
}
