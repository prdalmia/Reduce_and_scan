#include "hip/hip_runtime.h"
// Author: Nic Olsen

#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

#include "scan.cuh"
#include <hip/hip_cooperative_groups.h>
namespace cg = cooperative_groups;
// Scans each block of g_idata separately and writes the result to g_odata.
// g_idata and g_odata are arrays available on device of length n
// Writes the sum of each block to lasts[blockIdx.x]


__global__ void hillis_steele(float* g_odata, float* lasts,  float* g_idata, unsigned int n, bool write_lasts) {
    extern volatile __shared__ float s[];
    
    float *tmp1;
    float * tmp2;
    float* tmp3;
    bool write_p = write_lasts;
    cg::grid_group grid = cg::this_grid(); 
    int a = n;
    int tid = threadIdx.x;
    unsigned int index = blockDim.x * blockIdx.x + tid;
    int pout = 0;
    int pin = 1;
   for( int i = 0 ; i < 2 ; i++){
       pout = 0;
       pin = 1;
    if (index >= a) {
        s[tid] = 0.f;
    } else if (tid == 0) {
        s[tid] = 0.f;
    } else {
        s[tid] = g_idata[index - 1];
       }
    
    __syncthreads();

    for (unsigned int offset = 1; offset < blockDim.x; offset <<= 1) {
        pout = 1 - pout;
        pin = 1 - pout;

        if (tid >= offset) {
            s[pout * blockDim.x + tid] = s[pin * blockDim.x + tid] + s[pin * blockDim.x + tid - offset];
        } else {
            s[pout * blockDim.x + tid] = s[pin * blockDim.x + tid];
        }
        __syncthreads();
    }
    if (index < a ) {
        g_odata[index] = s[pout * blockDim.x + tid];
    }

    if (write_p && threadIdx.x == 0) {
        unsigned int block_end = blockIdx.x * blockDim.x + blockDim.x - 1;
        lasts[blockIdx.x] = s[pout * blockDim.x + blockDim.x - 1] + g_idata[block_end];
    }
    __syncthreads();
    cg::sync(grid); 
    if(a == n){
      tmp1 = g_idata;
      tmp2 = g_odata;
      tmp3 = lasts;
      g_idata = lasts;
      g_odata = lasts;
      lasts = nullptr;
      write_p = false;
      a = (n + blockDim.x - 1) / blockDim.x;
    }
   }
   cg::sync(grid); 
    lasts = tmp3;
    g_odata = tmp2;
    __syncthreads();
    if (index < n) {
        g_odata[index] = g_odata[index] + lasts[blockIdx.x];
      //  printf("g_odata is %f at index %d\n", g_odata[index], index);
    }
}

// Increment each element corresponding to block b_i of arr by lasts[b_i]
__global__ void inc_blocks(float* arr, float* lasts, unsigned int n) {
    unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < n) {
        arr[index] = arr[index] + lasts[blockIdx.x];
    }
}


__host__ void scan( float* in, float* out, unsigned int n, unsigned int threads_per_block) {
    // Sort each block indiviually
    unsigned int nBlocks = (n + threads_per_block - 1) / threads_per_block;
    float* lasts;
    hipMallocManaged(&lasts, nBlocks * sizeof(float));
    bool write_lasts = true;
    unsigned int shmem = 4 * threads_per_block * sizeof(float);
   // hillis_steele<<<nBlocks, threads_per_block, shmem>>>(out, lasts, in, n, true);
    //hipDeviceSynchronize();
   //for (unsigned int a = n; a > 1; a = (a + threads_per_block - 1) / threads_per_block) {
    void *kernelArgs[] = {
        (void *)&out,  (void *)&lasts, (void *)&in, (void *)&n, (void *)&write_lasts  
    };
    hipLaunchCooperativeKernel((void*)hillis_steele, nBlocks, threads_per_block,  kernelArgs, shmem, 0);
    //hillis_steele<<<nBlocks, threads_per_block, shmem>>>(out, lasts, in, n, true);
    // Swap input and output arrays
 //   float* tmp = in;
 //   in = lasts;
 //   lasts = tmp;
 //   std::cout << in[a-1] << std::endl;
 //  }
    // Scan lasts
    //hillis_steele<<<1, threads_per_block, shmem>>>(lasts, nullptr, lasts, nBlocks, false);
    hipDeviceSynchronize();

    hipFree(lasts);
}
