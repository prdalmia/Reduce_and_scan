#include "hip/hip_runtime.h"
// Author: Nic Olsen

#include <hip/hip_runtime.h>
#include <iostream>

#include "scan.cuh"

// Scans each block of g_idata separately and writes the result to g_odata.
// g_idata and g_odata are arrays available on device of length n
// Writes the sum of each block to lasts[blockIdx.x]
__global__ void hillis_steele(float* g_odata, float* lasts, const float* g_idata, unsigned int n, bool write_lasts) {
    extern volatile __shared__ float s[];

    int tid = threadIdx.x;
    unsigned int index = blockDim.x * blockIdx.x + tid;
    int pout = 0;
    int pin = 1;

    if (index >= n) {
        s[tid] = 0.f;
    } else if (tid == 0) {
        s[tid] = 0.f;
    } else {
        s[tid] = g_idata[index - 1];
    }
    
    __syncthreads();

    for (unsigned int offset = 1; offset < blockDim.x; offset <<= 1) {
        pout = 1 - pout;
        pin = 1 - pout;

        if (tid >= offset) {
            s[pout * blockDim.x + tid] = s[pin * blockDim.x + tid] + s[pin * blockDim.x + tid - offset];
        } else {
            s[pout * blockDim.x + tid] = s[pin * blockDim.x + tid];
        }
        __syncthreads();
    }
    if (index < n) {
        g_odata[index] = s[pout * blockDim.x + tid];
        if(gridDim.x == 1)
        printf("g_odata is %f at index %d\n", g_odata[index], index);
    }

    if (write_lasts && threadIdx.x == 0) {
        unsigned int block_end = blockIdx.x * blockDim.x + blockDim.x - 1;
        lasts[blockIdx.x] = s[pout * blockDim.x + blockDim.x - 1] + g_idata[block_end];
    }
}

// Increment each element corresponding to block b_i of arr by lasts[b_i]
__global__ void inc_blocks(float* arr, const float* lasts, unsigned int n) {
    unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < n) {
        arr[index] = arr[index] + lasts[blockIdx.x];
    }
}

__host__ void scan(float* in, float* out, unsigned int n, unsigned int threads_per_block) {
    // Sort each block indiviually
    unsigned int nBlocks = (n + threads_per_block - 1) / threads_per_block;
    float* lasts;
    hipMallocManaged(&lasts, nBlocks * sizeof(float));
    unsigned int shmem = 2 * threads_per_block * sizeof(float);
    hillis_steele<<<nBlocks, threads_per_block, shmem>>>(out, lasts, in, n, true);
    hipDeviceSynchronize();

    // Scan lasts
    
    hillis_steele<<<1, threads_per_block, shmem>>>(lasts, nullptr, lasts, nBlocks, false);
    hipDeviceSynchronize();

    // Add starting value to each block
    inc_blocks<<<nBlocks, threads_per_block>>>(out, lasts, n);
    hipDeviceSynchronize();

    hipFree(lasts);
}
